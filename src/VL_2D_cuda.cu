
#include <hip/hip_runtime.h>
/*! \file VL_2D_cuda.cu
 *  \brief Definitions of the cuda 2D VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"VL_2D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_vl_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_vl_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_2D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_2D.h"


__global__ void Update_Conserved_Variables_2D_half(Real *dev_conserved, Real *dev_conserved_half, 
                                                   Real *dev_F_x, Real *dev_F_y, int nx, int ny,
                                                   int n_ghost, Real dx, Real dy, Real dt, Real gamma);


Real VL_Algorithm_2D_CUDA(Real *host_conserved, int nx, int ny, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real dt)
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array

  #ifdef TIME
  // capture the start time
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  float elapsedTime;
  #endif

  int n_fields = 5;
  #ifdef DE
  n_fields++;
  #endif

  // dimensions of subgrid blocks
  int nx_s; //number of cells in the subgrid block along x direction
  int ny_s; //number of cells in the subgrid block along y direction
  int nz_s = 1; //number of cells in the subgrid block along z direction
  int x_off_s, y_off_s; // x and y offsets for subgrid block

  // total number of blocks needed
  int block_tot;    //total number of subgrid blocks (unsplit == 1)
  int block1_tot;   //total number of subgrid blocks in x direction
  int block2_tot;   //total number of subgrid blocks in y direction
  int remainder1;   //modulus of number of cells after block subdivision in x direction
  int remainder2;   //modulus of number of cells after block subdivision in y direction 

  // counter for which block we're on
  int block = 0;

  // calculate the dimensions for each subgrid block
  sub_dimensions_2D(nx, ny, n_ghost, &nx_s, &ny_s, &block1_tot, &block2_tot, &remainder1, &remainder2, n_fields);
  printf("%d %d %d %d %d %d\n", nx_s, ny_s, block1_tot, block2_tot, remainder1, remainder2);
  block_tot = block1_tot*block2_tot;

  // number of cells in one subgrid block
  int BLOCK_VOL = nx_s*ny_s*nz_s;

  // define the dimensions for the 2D grid
  int  ngrid = (BLOCK_VOL + 2*TPB - 1) / (2*TPB);

  //number of blocks per 2-d grid  
  dim3 dim2dGrid(ngrid, 2, 1);

  //number of threads per 1-d block   
  dim3 dim1dBlock(TPB, 1, 1);

  // allocate buffer arrays to copy conserved variable slices into
  Real **buffer;
  allocate_buffers_2D(block1_tot, block2_tot, BLOCK_VOL, buffer, n_fields);
  // and set up pointers for the location to copy from and to
  Real *tmp1;
  Real *tmp2;

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(2*ngrid*sizeof(Real));

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved, *dev_conserved_half;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *F_x, *F_y;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *etah_x, *etah_y;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;


  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, 2*ngrid*sizeof(Real)) );
  

  // transfer first conserved variable slice into the first buffer
  host_copy_init_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, remainder1, BLOCK_VOL, host_conserved, buffer, &tmp1, &tmp2, n_fields);
  
  // START LOOP OVER SUBGRID BLOCKS HERE
  while (block < block_tot) {

    // calculate the global x and y offsets of this subgrid block
    // (only needed for gravitational potential)
    get_offsets_2D(nx_s, ny_s, n_ghost, x_off, y_off, block, block1_tot, block2_tot, remainder1, remainder2, &x_off_s, &y_off_s);    


    // zero all the GPU arrays
    cudaMemset(dev_conserved, 0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(dev_conserved_half, 0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Lx,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Rx,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Ly,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Ry,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(F_x,   0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(F_y,   0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(eta_x,  0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(eta_y,  0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(etah_x, 0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(etah_y, 0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(dev_dti_array, 0, 2*ngrid*sizeof(Real));
    CudaCheckError();

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );


    // Step 1: Use PCM reconstruction to put conserved variables into interface arrays
    PCM_Reconstruction_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();


    // Step 2: Calculate first-order upwind fluxes 
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    #ifdef HLLC 
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    CudaCheckError();


    // Step 3: Update the conserved variables half a timestep 
    Update_Conserved_Variables_2D_half<<<dim2dGrid,dim1dBlock>>>(dev_conserved, dev_conserved_half, F_x, F_y, nx_s, ny_s, n_ghost, dx, dy, 0.5*dt, gama);
    CudaCheckError();


    // Step 4: Construct left and right interface values using updated conserved variables
    #ifdef PLMP
    PLMP_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    PLMP_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef PLMC
    PLMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PLMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);    
    #endif
    #ifdef PPMP
    PPMP_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    PPMP_VL<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif //PPMP
    #ifdef PPMC
    PPMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PPMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif //PPMC
    CudaCheckError();


    #ifdef H_CORRECTION
    // Step 4.5: Calculate eta values for H correction
    calc_eta_x_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, n_ghost, gama);
    calc_eta_y_2D<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_x, nx_s, ny_s, n_ghost);
    calc_etah_y_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_y, nx_s, ny_s, n_ghost);
    CudaCheckError();
    #endif


    // Step 5: Calculate the fluxes again
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    #ifdef HLLC 
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    CudaCheckError();


    // Step 6: Update the conserved variable array
    Update_Conserved_Variables_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, nx_s, ny_s, x_off_s, y_off_s, n_ghost, dx, dy, xbound, ybound, dt, gama);
    CudaCheckError();


    #ifdef DE
    Sync_Energies_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, gama);
    #endif        


    // Apply cooling
    #ifdef COOLING_GPU
    printf("Need to fix cooling.\n");
    //cooling_kernel<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, dt, gama);
    #endif


    // Step 7: Calculate the next timestep
    Calc_dt_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, dx, dy, dev_dti_array, gama);
    CudaCheckError();  


    // copy the conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );

    // copy the next conserved variable blocks into appropriate buffers
    host_copy_next_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved, buffer, &tmp1, n_fields);

    // copy the updated conserved variable array back into the host_conserved array on the CPU
    host_return_values_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved, buffer, n_fields);


    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, 2*ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the maximum inverse dt for this subgrid block
    for (int i=0; i<2*ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }


    // add one to the counter
    block++;

  }


  // free the CPU memory
  free(host_dti_array);
  free_buffers_2D(nx, ny, nx_s, ny_s, block1_tot, block2_tot, buffer);

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(dev_dti_array);


  // return the maximum inverse timestep
  return max_dti;

}


__global__ void Update_Conserved_Variables_2D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, Real gamma)
{
  int id, xid, yid, n_cells;
  int imo, jmo;

  Real dtodx = dt/dx;
  Real dtody = dt/dy;

  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;


  // all threads but one outer ring of ghost cells 
  if (xid > 0 && xid < nx-1 && yid > 0 && yid < ny-1)
  {
    // update the conserved variable array
    imo = xid-1 + yid*nx;
    jmo = xid + (yid-1)*nx;
    dev_conserved_half[            id] = dev_conserved[            id] 
                                       + dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                       + dtody * (dev_F_y[            jmo] - dev_F_y[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id] 
                                       + dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]) 
                                       + dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id] 
                                       + dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]) 
                                       + dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]); 
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id] 
                                       + dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                       + dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id] 
                                       + dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                       + dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
  } 
}




#endif //VL
#endif //CUDA

